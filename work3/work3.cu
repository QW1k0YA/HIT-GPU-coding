#include <stdio.h>
#include <hip/hip_runtime.h>
#include <iostream>
#include "vector"
#define N 16
#define BLOCKSIZE 2
using namespace std;

__global__ void muiltiply_shared_mem(double *A,double *B,double *C,int width)
{
    /// a block is a submatrix
    __shared__ double A1[BLOCKSIZE* BLOCKSIZE];
    __shared__ double B1[BLOCKSIZE* BLOCKSIZE];

    int x_index = threadIdx.x + blockDim.x*blockIdx.x;
    int y_index = threadIdx.y + blockDim.y*blockIdx.y;

    ///load data
    double sum_ = 0;
    for(int i = 0;i < width/BLOCKSIZE;i++)
    {
        A1[threadIdx.y*BLOCKSIZE + threadIdx.x] = A[y_index*width + threadIdx.x + i * BLOCKSIZE];
        B1[threadIdx.y*BLOCKSIZE + threadIdx.x] = B[width*(threadIdx.y + i*BLOCKSIZE) + x_index];
        /// load all data in the submatrix
        __syncthreads();


        ///for each thread
        for(int j = 0;j < BLOCKSIZE;j++)
        {
            sum_ += A1[threadIdx.y*BLOCKSIZE + j]*B1[j*BLOCKSIZE + threadIdx.x];
        }

        __syncthreads();
        ///after calculate ,then load the next buffer

    }

    C[y_index*width + x_index] = sum_;

}
int main()
{
    int n =N;
    double *dev_A,*dev_B,*dev_C;
    double A[N],B[N],C[N];

    for(int i = 0;i < n;i ++)
    {
        A[i] = i;
        B[i] = i*i%8;
    }

    hipMalloc(&dev_A, n * sizeof(double));
    hipMalloc(&dev_B, n * sizeof(double));
    hipMalloc(&dev_C, n * sizeof(double));

    hipMemcpy(dev_A,A,n*sizeof(double),hipMemcpyHostToDevice);
    hipMemcpy(dev_B,B,n*sizeof(double),hipMemcpyHostToDevice);

    ///<<<numBlocks, blockSize>>>
    ///choose the size
    dim3 blockdim(BLOCKSIZE,BLOCKSIZE);
    dim3 griddim(sqrt(n)/BLOCKSIZE,sqrt((n)/BLOCKSIZE));

    double t1,t2;
    t1 = clock();
    muiltiply_shared_mem<<<griddim,blockdim>>>(dev_A,dev_B,dev_C, sqrt(n));
    t2 = clock();

    hipMemcpy(C,dev_C,n*sizeof(double),hipMemcpyDeviceToHost);

    cout << "A:\n";
    int width = sqrt(n);

    for(int ii = 0; ii < n;ii += width)
    {
        for(int i = ii;i < ii + width;i ++)
        {
            cout << A[i] << " ";
        }
        cout << "\n";
    }


    cout << "\nB:\n";
    for(int ii = 0; ii < n;ii += width)
    {
        for(int i = ii;i < ii + width;i ++)
        {
            cout << B[i] << " ";
        }
        cout << "\n";
    }

    cout << "\nC:\n";
    for(int ii = 0; ii < n;ii += width)
    {
        for(int i = ii;i < ii + width;i ++)
        {
            cout << C[i] << " ";
        }
        cout << "\n";
    }

    cout <<(t2 - t1)/CLOCKS_PER_SEC<<"secs" << endl;

}

